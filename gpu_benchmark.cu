// File: gpu_benchmark.cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define NUM_SIZES 5

int main() {
    int sizes[NUM_SIZES] = {256, 512, 1024, 2048, 4096};

    // Open the output file
    FILE *output_file = fopen("gpu_benchmark_results.csv", "w");
    if (output_file == NULL) {
        printf("Error opening output file!\n");
        return -1;
    }

    // Write CSV header
    fprintf(output_file, "NumGPUs,MatrixSize,TimeSeconds\n");

    // Check number of devices
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    printf("Number of CUDA devices: %d\n", device_count);

    for(int s=0; s<NUM_SIZES; s++) {
        int N = sizes[s];

        // Allocate host matrices
        float *h_A = (float*)malloc(N*N*sizeof(float));
        float *h_B = (float*)malloc(N*N*sizeof(float));
        float *h_C = (float*)malloc(N*N*sizeof(float));

        // Initialize matrices with random numbers
        srand(0); // For reproducibility
        for(int i=0; i<N*N; i++) {
            h_A[i] = (float)rand() / RAND_MAX;
            h_B[i] = (float)rand() / RAND_MAX;
            h_C[i] = 0.0f;
        }

        // Ensure data is not cached
        for(int i=0; i<N*N; i++) {
            volatile float tmp = h_A[i] + h_B[i];
        }

        // Single GPU computation
        {
            hipSetDevice(0);

            // Allocate device matrices
            float *d_A, *d_B, *d_C;
            hipMalloc((void**)&d_A, N*N*sizeof(float));
            hipMalloc((void**)&d_B, N*N*sizeof(float));
            hipMalloc((void**)&d_C, N*N*sizeof(float));

            // Copy matrices to device
            hipMemcpy(d_A, h_A, N*N*sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, N*N*sizeof(float), hipMemcpyHostToDevice);

            // Ensure data is not cached
            hipDeviceSynchronize();

            // Use cuBLAS sgemm
            hipblasHandle_t handle;
            hipblasCreate(&handle);
            float alpha = 1.0f;
            float beta = 0.0f;

            hipDeviceSynchronize();
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        N, N, N, &alpha,
                        d_B, N,
                        d_A, N,
                        &beta,
                        d_C, N);

            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);

            // Write the result to the output file
            fprintf(output_file, "%d,%d,%f\n", 1, N, elapsedTime / 1000.0f);

            // Cleanup
            hipblasDestroy(handle);
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }

        // Multi-GPU computation (if device_count >= 2)
        if(device_count >= 2) {
            int num_devices = 2;
            int N_per_device = N / num_devices;

            // Allocate device matrices on each GPU
            float *d_A[2], *d_B[2], *d_C[2];
            hipblasHandle_t handles[2];

            for(int d=0; d<num_devices; d++) {
                hipSetDevice(d);
                int start_row = d * N_per_device;
                int rows = (d == num_devices - 1) ? N - start_row : N_per_device;

                hipMalloc((void**)&d_A[d], N*rows*sizeof(float));
                hipMalloc((void**)&d_B[d], N*N*sizeof(float));
                hipMalloc((void**)&d_C[d], N*rows*sizeof(float));

                // Copy relevant parts of matrices to each device
                hipMemcpy(d_A[d], h_A + start_row*N, N*rows*sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(d_B[d], h_B, N*N*sizeof(float), hipMemcpyHostToDevice);

                hipblasCreate(&handles[d]);
            }

            // Ensure data is not cached
            hipDeviceSynchronize();

            // Start timing
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start, 0);

            // Launch computations on each device
            for(int d=0; d<num_devices; d++) {
                hipSetDevice(d);

                int start_row = d * N_per_device;
                int rows = (d == num_devices - 1) ? N - start_row : N_per_device;

                float alpha = 1.0f;
                float beta = 0.0f;

                hipblasSetStream(handles[d], 0);

                hipblasSgemm(handles[d], HIPBLAS_OP_N, HIPBLAS_OP_N,
                            N, rows, N, &alpha,
                            d_B[d], N,
                            d_A[d], N,
                            &beta,
                            d_C[d], N);
            }

            // Synchronize devices
            for(int d=0; d<num_devices; d++) {
                hipSetDevice(d);
                hipDeviceSynchronize();
            }

            // End timing
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);

            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);

            // Write the result to the output file
            fprintf(output_file, "%d,%d,%f\n", num_devices, N, elapsedTime / 1000.0f);

            // Cleanup
            for(int d=0; d<num_devices; d++) {
                hipblasDestroy(handles[d]);
                hipFree(d_A[d]);
                hipFree(d_B[d]);
                hipFree(d_C[d]);
            }
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }

        free(h_A);
        free(h_B);
        free(h_C);
    }

    fclose(output_file);

    return 0;
}
